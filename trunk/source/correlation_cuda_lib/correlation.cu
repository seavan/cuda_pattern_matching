#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <dos.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

extern __shared__ int sdata[];

#define PATNDATA(index) sdata[index]
#define PATNDATAY(index) sdata[index + 27]
#define KERNELDATAX(index) PATNDATA(index + 9)
#define KERNELDATAY(index) PATNDATA(index + 18)

#define SHAREDSIZE 300

#define uchar unsigned char


__global__ void
correlation( uchar* g_idata, uchar* g_pdata, uchar* g_odata, int _width, int _height, int _ptnWidth, int _ptnHeight, int i, int j) 
{
	// ���������� �������� ��������������� ��������
	int x = (blockIdx.x) * blockDim.x + threadIdx.x + i;
	int y = (blockIdx.y) * blockDim.y + threadIdx.y + j;

	// ��������� �������
	if( (x >= _width) || (y >= _height) )
	{
		return;
	}

	// ���������� �������� ��� ������ ����������
	uchar* curResult = g_odata + y * _width + x;

	// ��������� �������
	if( (x > _width - _ptnWidth) || (y > _height - _ptnHeight) )
	{
		*curResult = 0xFF;
		return;
	}
	
	// ������ ��������� ��������� ��� �������
	uchar* curPattern = g_pdata;

	// ��������� �������� ����������
	float result = 0;

	// ����� ��������� �������� ��� ����������� � �������
	float nom1 = 0;
	float nom2 = 0;

	// ���������� ����������
	for(int i = 0; i < _ptnHeight; ++i)
	{
		uchar* curRow = g_idata + x + (y + i) * _width;
		for(int j = 0; j < _ptnWidth; ++j)
		{	
			// ������� ������� ����������� � �������
			uchar cv = *curRow;
			uchar cp = *curPattern;	
			
			// ��������
			int delta = cv - cp;

			// ��������� ����������
			result += delta * delta;
			
			// ������� ������� � �����������
			nom1 += cv * cv;
			nom2 += cp * cp;

			// ������� ������
			++curRow;
			++curPattern;
		}
	}

	// ���������� ���������� ������������ ���������� (1 - �����)
	result /= sqrt(nom1 * nom2);

	// ������������� �� ���� 255 * 4
	result *= 1024;

	// ������ ������������ ��������
	result = result > 255 ? 255 : result;

	// ������� ���������
	*curResult = (uchar)(result);
}

void
correlationCpu( uchar* g_idata, uchar* g_pdata, uchar* g_odata, int _width, int _height, int _ptnWidth, int _ptnHeight, int i, int j) 
{
	int x = i;//(blockIdx.x) * blockDim.x + threadIdx.x + i;
	int y = j;//(blockIdx.y) * blockDim.y + threadIdx.y + j;

	if( (x >= _width) || (y >= _height) )
	{
		return;
	}

	uchar* curResult = g_odata + y * _width + x;

	if( (x > _width - _ptnWidth) || (y > _height - _ptnHeight) )
	{
		*curResult = 0xFF;
		return;
	}

	uchar* curPattern = g_pdata;
	float result = 0;
	float nom1 = 0;
	float nom2 = 0;

	for(int i = 0; i < _ptnHeight; ++i)
	{
		uchar* curRow = g_idata + x + (y + i) * _width;
		for(int j = 0; j < _ptnWidth; ++j)
		{			
			uchar cv = *curRow;
			uchar cp = *curPattern;	
			int delta = cv - cp;
			result += delta * delta;
			//result += cv;
			nom1 += cv * cv;
			nom2 += cp * cp;
			++curRow;
			++curPattern;
		}
	}

	result /= sqrt(nom1 * nom2);
	result *= 1024; // expanding value range
	result = result > 255 ? 255 : result;
	*curResult = (uchar)(result);
}


extern "C" unsigned char* correlationCpu( unsigned char* _input, int _width, int _height, unsigned char* _pattern, int _ptnwidth, int _ptnheight)
{
	uchar* fResult = new uchar[_width * _height];

	for(int i = 0; i < _width; ++i)
		for(int j = 0; j < _height; ++j)
		{
			correlationCpu( _input, _pattern, fResult, _width, _height, _ptnwidth, _ptnheight, i, j);
		}

	return fResult;
}

extern "C" unsigned char* correlationCuda( int argc, char** argv, 
								   unsigned char* _input, int _width, int _height, unsigned char* _pattern, int _ptnwidth, int _ptnheight)
{
	// ���������������� ����������
	CUT_DEVICE_INIT(argc, argv);

	// ���������������� ������
    unsigned int timer = 0;
    CUT_SAFE_CALL( cutCreateTimer( &timer));
    CUT_SAFE_CALL( cutStartTimer( timer));

    // ���������� � �������� ������
	int imsize = _width * _height;
	int ptnsize = _ptnwidth * _ptnheight;
	uchar* fImage = _input;
	uchar* fPattern = _pattern;

	// �������� ������ ��� �����������
	uchar* d_idata = NULL;
	printf("Allocating device memory for image\n");
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_idata, imsize * sizeof(unsigned char)));
	printf("Address: %d\n", d_idata);

	// �������� ������ ��� �������
	uchar* d_pdata = NULL;
	printf("Allocating device memory for pattern\n");
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_pdata, ptnsize * sizeof(unsigned char)));
	printf("Address: %d\n", d_pdata);

	// ��������� �����������
    CUDA_SAFE_CALL( hipMemcpy( d_idata, fImage, imsize * sizeof(unsigned char),
                                hipMemcpyHostToDevice) );
	CUT_CHECK_ERROR("Image loading failed\n");

	// ��������� ������
	CUDA_SAFE_CALL( hipMemcpy( d_pdata, fPattern, ptnsize * sizeof(unsigned char),
                                hipMemcpyHostToDevice) );   
	CUT_CHECK_ERROR("Pattern loading failed\n");

	// �������� ������ ��� ����������
    uchar* d_odata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata, imsize * sizeof(unsigned char) ));
	printf("Address: %d\n", d_odata);

	// ���������� ������ ����� �������
	#define BLOCK_SIZE 4
	#define THREAD_SIZE_X 16
	#define THREAD_SIZE_Y 16

	// ������ ��������� ������ ����
    dim3  grid( BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3  threads( THREAD_SIZE_X, THREAD_SIZE_Y, 1);

	// ���������� ������ ����
	int stepSizeX = (THREAD_SIZE_X * BLOCK_SIZE);
	int stepSizeY = (THREAD_SIZE_Y * BLOCK_SIZE);

	

	// ������� ���������� � ������ ���������
	printf("starting\n");

    // ��������� ������ �� �����������
	for(int i = 0; i < _width; i += stepSizeX)
		for(int j = 0; j < _height; j += stepSizeY)
		{
			// ������� ���� (16 ������ �� 256 �������)
			correlation<<< grid, threads >>>( d_idata, d_pdata, d_odata, _width, _height, _ptnwidth, _ptnheight, i, j);
			hipDeviceSynchronize();
		}   

    // ��������� ����� ����������
    CUT_CHECK_ERROR("Kernel execution failed");

    // ��������� ������ � ���������� � ������
	uchar* fResult = (uchar*)malloc(imsize * sizeof(unsigned char));
    CUDA_SAFE_CALL( hipMemcpy( fResult, d_odata, imsize,
                                hipMemcpyDeviceToHost) );
	CUT_CHECK_ERROR("Output copy failed");

	// ���������� ������, ������� ������ �������
    CUT_SAFE_CALL( cutStopTimer( timer)); 
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    CUT_SAFE_CALL( cutDeleteTimer( timer));
	
	// ������� ���������
	return fResult;
}

